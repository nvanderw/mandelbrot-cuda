
#include <hip/hip_runtime.h>
#include <stdio.h>
#define ABS(X) X < 0 ? -X : X


extern "C" float *create_mandelbrot(int res_x, int res_y, float min_x, float min_y,
        float max_x, float max_y, int iter);

__global__ void mandelbrot(float *region, int offset, int2 res, float4 boundary, int iter);


/* We define a maximum number of iterations the kernel can do, so that
 * it isn't rudely interrupted by the watchdog timer */
static const long max_iterations = 1L<<30;
static const int block_size = 128;

extern "C" float *create_mandelbrot(int res_x, int res_y, float min_x, float min_y,
        float max_x, float max_y, int iter) {

    float *region, *d_region;
    dim3 threadsPerBlock(block_size);
    dim3 blocks;
    float4 boundary;

    int size = res_x * res_y;
    size_t host_bytes = (size_t) size * sizeof(float);
    size_t job_bytes;
    size_t jobsize;
    hipStream_t copy_stream; // Stream for copying between host and device
    if(hipStreamCreate(&copy_stream) != hipSuccess)
        return NULL;

    region = (float*) malloc(host_bytes);
    if(!region)
        goto cleanup3;

    /* Find some number of pixels that is a power of two, evenly divides up
     * the work, and causes the kernel to compute at most max_iterations */

    jobsize = block_size;
    for(;!(size & (jobsize - 1)) && (jobsize * iter <= max_iterations); jobsize <<= 1);
    jobsize >>= 1;
    if(jobsize < block_size)
        goto cleanup3;

    fprintf(stderr, "%s: %d. %s: %d.\n", "Number of pixels", size,
            "Number of pixels per job", jobsize);

    blocks = dim3(jobsize / threadsPerBlock.x);
    job_bytes = jobsize * sizeof(float);

    boundary.x = min_x;
    boundary.y = min_y;
    boundary.z = max_x;
    boundary.w = max_y;

    if(hipMalloc(&d_region, job_bytes) != hipSuccess)
        goto cleanup2;


    for(int start = 0; start <= (size - jobsize); start += jobsize) {
        mandelbrot<<<blocks, threadsPerBlock>>>(d_region, start, make_int2(res_x, res_y), boundary, iter);
        if(hipMemcpy(&region[start], d_region, job_bytes, hipMemcpyDeviceToHost) != hipSuccess)
            goto cleanup1;
    }

    hipFree(d_region);
    hipStreamDestroy(copy_stream);
    return region;

cleanup1:
    hipFree(d_region);
cleanup2:
    free(region);
cleanup3:
    hipStreamDestroy(copy_stream);
    return NULL;
}


/* Calculates the mandelbrot set and stores the results in region, which
 * should be of length gridDim.x * blockDim.x * gridDim.y * blockDim.y
 * The float4 boundary specifies the region of the complex plane to test
 * for divergence, with the x, y, z, and w components representing the minimum
 * x, minimum y, maximum x, and maximum y values of the rectangular region. */
__global__ void mandelbrot(float *region, int offset, int2 res, float4 boundary, int iter) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel = index + offset;
    int pixel_x = pixel % res.x;
    int pixel_y = pixel / res.x;

    double2 c, z;
    int i;

    c.x = ((boundary.z - boundary.x) * (((double) pixel_x) + 0.5))/(res.x) + boundary.x;
    c.y = ((boundary.y - boundary.w) * (((double) pixel_y) + 0.5))/(res.y) + boundary.w;
    z = c;

    for(i=1;(i<iter) && (z.x*z.x+z.y*z.y <= 4);i++)
        z = make_double2(z.x * z.x - z.y*z.y + c.x,
                         2*z.x*z.y + c.y);

    region[index] = (i >= iter) ? (-1) : i + 2 - log2f(log2f(z.x*z.x+z.y*z.y));
}
